
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void mykernel(void) 
{
printf("Hello World from GPU!\n");
}


int main(void) 
{
mykernel<<<1,1>>>();
hipDeviceSynchronize;
printf("Hello World from CPU!\n");

return 0;
}